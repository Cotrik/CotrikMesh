#include "hip/hip_runtime.h"
/* =========================================================================
   Copyright (c) 2010-2016, Institute for Microelectronics,
                            Institute for Analysis and Scientific Computing,
                            TU Wien.
   Portions of this software are copyright by UChicago Argonne, LLC.

                            -----------------
                  ViennaCL - The Vienna Computing Library
                            -----------------

   Project Head:    Karl Rupp                   rupp@iue.tuwien.ac.at

   (A list of authors and contributors can be found in the PDF manual)

   License:         MIT (X11), see file LICENSE in the base directory
============================================================================= */


/*
*   Benchmark:  Sparse matrix operations, i.e. matrix-vector products (sparse.cpp and sparse.cu are identical, the latter being required for compilation using CUDA nvcc)
*
*/

//#define VIENNACL_BUILD_INFO
#ifndef NDEBUG
 #define NDEBUG
#endif

#define VIENNACL_WITH_UBLAS 1

#include <boost/numeric/ublas/triangular.hpp>
#include <boost/numeric/ublas/vector.hpp>
#include <boost/numeric/ublas/vector_proxy.hpp>
#include <boost/numeric/ublas/matrix_sparse.hpp>
#include <boost/numeric/ublas/operation_sparse.hpp>
#include <boost/numeric/ublas/lu.hpp>


#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/coordinate_matrix.hpp"
#include "viennacl/compressed_matrix.hpp"
#include "viennacl/ell_matrix.hpp"
#include "viennacl/hyb_matrix.hpp"
#include "viennacl/sliced_ell_matrix.hpp"
#include "viennacl/linalg/prod.hpp"
#include "viennacl/linalg/norm_2.hpp"
#include "viennacl/io/matrix_market.hpp"
#include "viennacl/linalg/ilu.hpp"
#include "viennacl/tools/timer.hpp"


#include <iostream>
#include <vector>


#define BENCHMARK_RUNS          10


inline void printOps(double num_ops, double exec_time)
{
  std::cout << "GFLOPs: " << num_ops / (1000000 * exec_time * 1000) << std::endl;
}


template<typename ScalarType>
int run_benchmark()
{
  viennacl::tools::timer timer;
  double exec_time;

  ScalarType std_factor1 = ScalarType(3.1415);
  ScalarType std_factor2 = ScalarType(42.0);
  viennacl::scalar<ScalarType> vcl_factor1(std_factor1);
  viennacl::scalar<ScalarType> vcl_factor2(std_factor2);

  boost::numeric::ublas::vector<ScalarType> ublas_vec1;
  boost::numeric::ublas::vector<ScalarType> ublas_vec2;

  boost::numeric::ublas::compressed_matrix<ScalarType> ublas_matrix;
  if (!viennacl::io::read_matrix_market_file(ublas_matrix, "../examples/testdata/mat65k.mtx"))
  {
    std::cout << "Error reading Matrix file" << std::endl;
    return 0;
  }
  //unsigned int cg_mat_size = cg_mat.size();
  std::cout << "done reading matrix" << std::endl;

  ublas_vec1 = boost::numeric::ublas::scalar_vector<ScalarType>(ublas_matrix.size1(), ScalarType(1.0));
  ublas_vec2 = ublas_vec1;

  viennacl::compressed_matrix<ScalarType, 1> vcl_compressed_matrix_1;
  viennacl::compressed_matrix<ScalarType, 4> vcl_compressed_matrix_4;
  viennacl::compressed_matrix<ScalarType, 8> vcl_compressed_matrix_8;

  viennacl::coordinate_matrix<ScalarType> vcl_coordinate_matrix_128;

  viennacl::ell_matrix<ScalarType, 1> vcl_ell_matrix_1;
  viennacl::hyb_matrix<ScalarType, 1> vcl_hyb_matrix_1;
  viennacl::sliced_ell_matrix<ScalarType> vcl_sliced_ell_matrix_1;

  viennacl::vector<ScalarType> vcl_vec1(ublas_vec1.size());
  viennacl::vector<ScalarType> vcl_vec2(ublas_vec1.size());

  //cpu to gpu:
  viennacl::copy(ublas_matrix, vcl_compressed_matrix_1);
  #ifndef VIENNACL_EXPERIMENTAL_DOUBLE_PRECISION_WITH_STREAM_SDK_ON_GPU
  viennacl::copy(ublas_matrix, vcl_compressed_matrix_4);
  viennacl::copy(ublas_matrix, vcl_compressed_matrix_8);
  #endif
  viennacl::copy(ublas_matrix, vcl_coordinate_matrix_128);
  viennacl::copy(ublas_matrix, vcl_ell_matrix_1);
  viennacl::copy(ublas_matrix, vcl_hyb_matrix_1);
  viennacl::copy(ublas_matrix, vcl_sliced_ell_matrix_1);
  viennacl::copy(ublas_vec1, vcl_vec1);
  viennacl::copy(ublas_vec2, vcl_vec2);


  ///////////// Matrix operations /////////////////

  std::cout << "------- Matrix-Vector product on CPU ----------" << std::endl;
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    //ublas_vec1 = boost::numeric::ublas::prod(ublas_matrix, ublas_vec2);
    boost::numeric::ublas::axpy_prod(ublas_matrix, ublas_vec2, ublas_vec1, true);
  }
  exec_time = timer.get();
  std::cout << "CPU time: " << exec_time << std::endl;
  std::cout << "CPU "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << ublas_vec1[0] << std::endl;


  std::cout << "------- Matrix-Vector product with compressed_matrix ----------" << std::endl;


  vcl_vec1 = viennacl::linalg::prod(vcl_compressed_matrix_1, vcl_vec2); //startup calculation
  vcl_vec1 = viennacl::linalg::prod(vcl_compressed_matrix_4, vcl_vec2); //startup calculation
  vcl_vec1 = viennacl::linalg::prod(vcl_compressed_matrix_8, vcl_vec2); //startup calculation
  //std_result = 0.0;

  viennacl::backend::finish();
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    vcl_vec1 = viennacl::linalg::prod(vcl_compressed_matrix_1, vcl_vec2);
  }
  viennacl::backend::finish();
  exec_time = timer.get();
  std::cout << "GPU time align1: " << exec_time << std::endl;
  std::cout << "GPU align1 "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << vcl_vec1[0] << std::endl;

  std::cout << "Testing triangular solves: compressed_matrix" << std::endl;

  viennacl::copy(ublas_vec1, vcl_vec1);
  viennacl::linalg::inplace_solve(trans(vcl_compressed_matrix_1), vcl_vec1, viennacl::linalg::unit_lower_tag());
  viennacl::copy(ublas_vec1, vcl_vec1);
  std::cout << "ublas..." << std::endl;
  timer.start();
  boost::numeric::ublas::inplace_solve(trans(ublas_matrix), ublas_vec1, boost::numeric::ublas::unit_lower_tag());
  std::cout << "Time elapsed: " << timer.get() << std::endl;
  std::cout << "ViennaCL..." << std::endl;
  viennacl::backend::finish();
  timer.start();
  viennacl::linalg::inplace_solve(trans(vcl_compressed_matrix_1), vcl_vec1, viennacl::linalg::unit_lower_tag());
  viennacl::backend::finish();
  std::cout << "Time elapsed: " << timer.get() << std::endl;

  ublas_vec1 = boost::numeric::ublas::prod(ublas_matrix, ublas_vec2);

  viennacl::backend::finish();
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    vcl_vec1 = viennacl::linalg::prod(vcl_compressed_matrix_4, vcl_vec2);
  }
  viennacl::backend::finish();
  exec_time = timer.get();
  std::cout << "GPU time align4: " << exec_time << std::endl;
  std::cout << "GPU align4 "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << vcl_vec1[0] << std::endl;

  viennacl::backend::finish();
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    vcl_vec1 = viennacl::linalg::prod(vcl_compressed_matrix_8, vcl_vec2);
  }
  viennacl::backend::finish();
  exec_time = timer.get();
  std::cout << "GPU time align8: " << exec_time << std::endl;
  std::cout << "GPU align8 "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << vcl_vec1[0] << std::endl;


  std::cout << "------- Matrix-Vector product with coordinate_matrix ----------" << std::endl;
  vcl_vec1 = viennacl::linalg::prod(vcl_coordinate_matrix_128, vcl_vec2); //startup calculation
  viennacl::backend::finish();

  viennacl::copy(vcl_vec1, ublas_vec2);
  long err_cnt = 0;
  for (std::size_t i=0; i<ublas_vec1.size(); ++i)
  {
    if ( fabs(ublas_vec1[i] - ublas_vec2[i]) / std::max(fabs(ublas_vec1[i]), fabs(ublas_vec2[i])) > 1e-2)
    {
      std::cout << "Error at index " << i << ": Should: " << ublas_vec1[i] << ", Is: " << ublas_vec2[i] << std::endl;
      ++err_cnt;
      if (err_cnt > 5)
        break;
    }
  }

  viennacl::backend::finish();
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    vcl_vec1 = viennacl::linalg::prod(vcl_coordinate_matrix_128, vcl_vec2);
  }
  viennacl::backend::finish();
  exec_time = timer.get();
  std::cout << "GPU time: " << exec_time << std::endl;
  std::cout << "GPU "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << vcl_vec1[0] << std::endl;


  std::cout << "------- Matrix-Vector product with ell_matrix ----------" << std::endl;
  vcl_vec1 = viennacl::linalg::prod(vcl_ell_matrix_1, vcl_vec2); //startup calculation
  viennacl::backend::finish();

  viennacl::copy(vcl_vec1, ublas_vec2);
  err_cnt = 0;
  for (std::size_t i=0; i<ublas_vec1.size(); ++i)
  {
    if ( fabs(ublas_vec1[i] - ublas_vec2[i]) / std::max(fabs(ublas_vec1[i]), fabs(ublas_vec2[i])) > 1e-2)
    {
      std::cout << "Error at index " << i << ": Should: " << ublas_vec1[i] << ", Is: " << ublas_vec2[i] << std::endl;
      ++err_cnt;
      if (err_cnt > 5)
        break;
    }
  }

  viennacl::backend::finish();
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    vcl_vec1 = viennacl::linalg::prod(vcl_ell_matrix_1, vcl_vec2);
  }
  viennacl::backend::finish();
  exec_time = timer.get();
  std::cout << "GPU time: " << exec_time << std::endl;
  std::cout << "GPU "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << vcl_vec1[0] << std::endl;


  std::cout << "------- Matrix-Vector product with hyb_matrix ----------" << std::endl;
  vcl_vec1 = viennacl::linalg::prod(vcl_hyb_matrix_1, vcl_vec2); //startup calculation
  viennacl::backend::finish();

  viennacl::copy(vcl_vec1, ublas_vec2);
  err_cnt = 0;
  for (std::size_t i=0; i<ublas_vec1.size(); ++i)
  {
    if ( fabs(ublas_vec1[i] - ublas_vec2[i]) / std::max(fabs(ublas_vec1[i]), fabs(ublas_vec2[i])) > 1e-2)
    {
      std::cout << "Error at index " << i << ": Should: " << ublas_vec1[i] << ", Is: " << ublas_vec2[i] << std::endl;
      ++err_cnt;
      if (err_cnt > 5)
        break;
    }
  }

  viennacl::backend::finish();
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    vcl_vec1 = viennacl::linalg::prod(vcl_hyb_matrix_1, vcl_vec2);
  }
  viennacl::backend::finish();
  exec_time = timer.get();
  std::cout << "GPU time: " << exec_time << std::endl;
  std::cout << "GPU "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << vcl_vec1[0] << std::endl;


  std::cout << "------- Matrix-Vector product with sliced_ell_matrix ----------" << std::endl;
  vcl_vec1 = viennacl::linalg::prod(vcl_sliced_ell_matrix_1, vcl_vec2); //startup calculation
  viennacl::backend::finish();

  viennacl::copy(vcl_vec1, ublas_vec2);
  err_cnt = 0;
  for (std::size_t i=0; i<ublas_vec1.size(); ++i)
  {
    if ( fabs(ublas_vec1[i] - ublas_vec2[i]) / std::max(fabs(ublas_vec1[i]), fabs(ublas_vec2[i])) > 1e-2)
    {
      std::cout << "Error at index " << i << ": Should: " << ublas_vec1[i] << ", Is: " << ublas_vec2[i] << std::endl;
      ++err_cnt;
      if (err_cnt > 5)
        break;
    }
  }

  viennacl::backend::finish();
  timer.start();
  for (int runs=0; runs<BENCHMARK_RUNS; ++runs)
  {
    vcl_vec1 = viennacl::linalg::prod(vcl_sliced_ell_matrix_1, vcl_vec2);
  }
  viennacl::backend::finish();
  exec_time = timer.get();
  std::cout << "GPU time: " << exec_time << std::endl;
  std::cout << "GPU "; printOps(2.0 * static_cast<double>(ublas_matrix.nnz()), static_cast<double>(exec_time) / static_cast<double>(BENCHMARK_RUNS));
  std::cout << vcl_vec1[0] << std::endl;

  return EXIT_SUCCESS;
}


int main()
{
  std::cout << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << "               Device Info" << std::endl;
  std::cout << "----------------------------------------------" << std::endl;

#ifdef VIENNACL_WITH_OPENCL
  std::cout << viennacl::ocl::current_device().info() << std::endl;
#endif
  std::cout << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << "## Benchmark :: Sparse" << std::endl;
  std::cout << "----------------------------------------------" << std::endl;
  std::cout << std::endl;
  std::cout << "   -------------------------------" << std::endl;
  std::cout << "   # benchmarking single-precision" << std::endl;
  std::cout << "   -------------------------------" << std::endl;
  run_benchmark<float>();
#ifdef VIENNACL_WITH_OPENCL
  if ( viennacl::ocl::current_device().double_support() )
#endif
  {
    std::cout << std::endl;
    std::cout << "   -------------------------------" << std::endl;
    std::cout << "   # benchmarking double-precision" << std::endl;
    std::cout << "   -------------------------------" << std::endl;
    run_benchmark<double>();
  }
  return 0;
}

