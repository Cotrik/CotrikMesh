#include "hip/hip_runtime.h"
/* =========================================================================
   Copyright (c) 2010-2016, Institute for Microelectronics,
                            Institute for Analysis and Scientific Computing,
                            TU Wien.
   Portions of this software are copyright by UChicago Argonne, LLC.

                            -----------------
                  ViennaCL - The Vienna Computing Library
                            -----------------

   Project Head:    Karl Rupp                   rupp@iue.tuwien.ac.at

   (A list of authors and contributors can be found in the PDF manual)

   License:         MIT (X11), see file LICENSE in the base directory
============================================================================= */

/** \example blas1.cpp
*
*   This tutorial shows how the BLAS level 1 functionality available in ViennaCL can be used.
*   Operator overloading in C++ is used extensively to provide an intuitive syntax.
*
*   We start off with including necessary headers:
**/


// include necessary system headers
#include <iostream>

//include basic scalar and vector types of ViennaCL
#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"

//include the generic inner product functions of ViennaCL
#include "viennacl/linalg/inner_prod.hpp"

//include the generic norm functions of ViennaCL
#include "viennacl/linalg/norm_1.hpp"
#include "viennacl/linalg/norm_2.hpp"
#include "viennacl/linalg/norm_inf.hpp"
#include "viennacl/tools/random.hpp"


/**
* In this tutorial we do not need additional auxiliary functions, allowing us to start right with main():
**/
int main()
{
  //Change this type definition to double if your gpu supports that
  typedef float       ScalarType;

  viennacl::tools::uniform_random_numbers<ScalarType> randomNumber;

  /**
  * <h2> Scalar Operations </h2>
  *
  * Although usually not very efficient because of PCI-Express latency, ViennaCL enables you to directly manipulate individual scalar values.
  * As such, a viennacl::scalar<double> behaves very similar to a normal `double`.
  *
  * Let us define a few CPU and ViennaCL scalars:
  *
  **/
  ScalarType s1 = ScalarType(3.1415926);   //note: writing ScalarType s1 = 3.1415926; leads to warnings with some compilers if ScalarType is 'float'.
  ScalarType s2 = ScalarType(2.71763);
  ScalarType s3 = ScalarType(42.0);

  viennacl::scalar<ScalarType> vcl_s1;
  viennacl::scalar<ScalarType> vcl_s2 = ScalarType(1.0);
  viennacl::scalar<ScalarType> vcl_s3 = ScalarType(1.0);

  /**
  * CPU scalars can be transparently assigned to GPU scalars and vice versa:
  **/
  std::cout << "Copying a few scalars..." << std::endl;
  vcl_s1 = s1;
  s2 = vcl_s2;
  vcl_s3 = s3;

  /**
  * Operations between GPU scalars work just as for CPU scalars:
  * (Note that such single compute kernels on the GPU are considerably slower than on the CPU)
  **/

  std::cout << "Manipulating a few scalars..." << std::endl;
  std::cout << "operator +=" << std::endl;
  s1 += s2;
  vcl_s1 += vcl_s2;

  std::cout << "operator *=" << std::endl;
  s1 *= s2;
  vcl_s1 *= vcl_s2;

  std::cout << "operator -=" << std::endl;
  s1 -= s2;
  vcl_s1 -= vcl_s2;

  std::cout << "operator /=" << std::endl;
  s1 /= s2;
  vcl_s1 /= vcl_s2;

  std::cout << "operator +" << std::endl;
  s1 = s2 + s3;
  vcl_s1 = vcl_s2 + vcl_s3;

  std::cout << "multiple operators" << std::endl;
  s1 = s2 + s3 * s2 - s3 / s1;
  vcl_s1 = vcl_s2 + vcl_s3 * vcl_s2 - vcl_s3 / vcl_s1;


  /**
  * Operations can also be mixed:
  **/
  std::cout << "mixed operations" << std::endl;
  vcl_s1 = s1 * vcl_s2 + s3 - vcl_s3;


  /**
  * The output stream is overloaded as well for direct printing to e.g. a terminal:
  **/

  std::cout << "CPU scalar s3: " << s3 << std::endl;
  std::cout << "GPU scalar vcl_s3: " << vcl_s3 << std::endl;


  /**
  *  <h2>Vector Operations</h2>
  *
  * Define a few vectors (from STL and plain C) and viennacl::vectors
  **/
  std::vector<ScalarType>      std_vec1(10);
  std::vector<ScalarType>      std_vec2(10);
  ScalarType                   plain_vec3[10];  //plain C array

  viennacl::vector<ScalarType> vcl_vec1(10);
  viennacl::vector<ScalarType> vcl_vec2(10);
  viennacl::vector<ScalarType> vcl_vec3(10);

  /**
  * Let us fill the CPU vectors with random values:
  **/

  for (unsigned int i = 0; i < 10; ++i)
  {
    std_vec1[i] = randomNumber();
    vcl_vec2(i) = randomNumber();  //also works for GPU vectors, but is MUCH slower (approx. factor 10.000) than the CPU analogue
    plain_vec3[i] = randomNumber();
  }

  /**
  * Copy the CPU vectors to the GPU vectors and vice versa
  **/
  viennacl::copy(std_vec1.begin(), std_vec1.end(), vcl_vec1.begin()); //either the STL way
  viennacl::copy(vcl_vec2.begin(), vcl_vec2.end(), std_vec2.begin()); //either the STL way
  viennacl::copy(vcl_vec2, std_vec2);                                 //using the short hand notation for objects that provide .begin() and .end() members
  viennacl::copy(vcl_vec2.begin(), vcl_vec2.end(), plain_vec3);       //copy to plain C vector

  /**
  * Also partial copies by providing the corresponding iterators are possible:
  **/
  viennacl::copy(std_vec1.begin() + 4, std_vec1.begin() + 8, vcl_vec1.begin() + 4);   //cpu to gpu
  viennacl::copy(vcl_vec1.begin() + 4, vcl_vec1.begin() + 8, vcl_vec2.begin() + 1);   //gpu to gpu
  viennacl::copy(vcl_vec1.begin() + 4, vcl_vec1.begin() + 8, std_vec1.begin() + 1);   //gpu to cpu

  /**
  * Compute the inner product of two GPU vectors and write the result to either CPU or GPU
  **/
  vcl_s1 = viennacl::linalg::inner_prod(vcl_vec1, vcl_vec2);
  s1 = viennacl::linalg::inner_prod(vcl_vec1, vcl_vec2);
  s2 = viennacl::linalg::inner_prod(std_vec1, std_vec2); //inner prod can also be used with std::vector (computations are carried out on CPU then)

  /**
  * Compute norms:
  **/
  s1 = viennacl::linalg::norm_1(vcl_vec1);
  vcl_s2 = viennacl::linalg::norm_2(vcl_vec2);
  s3 = viennacl::linalg::norm_inf(vcl_vec3);


  /**
  * Plane rotation of two vectors:
  **/
  viennacl::linalg::plane_rotation(vcl_vec1, vcl_vec2, 1.1f, 2.3f);

  /**
  * Use viennacl::vector via the overloaded operators just as you would write it on paper:
  **/

  //simple expression:
  vcl_vec1 = vcl_s1 * vcl_vec2 / vcl_s3;

  //more complicated expression:
  vcl_vec1 = vcl_vec2 / vcl_s3 + vcl_s2 * (vcl_vec1 - vcl_s2 * vcl_vec2);


  /**
  * Swap the content of two vectors without a temporary vector:
  **/
  viennacl::swap(vcl_vec1, vcl_vec2);  //swaps all entries in memory
  viennacl::fast_swap(vcl_vec1, vcl_vec2); //swaps OpenCL memory handles only

  /**
  * The vectors can also be cleared directly:
  **/
  vcl_vec1.clear();
  vcl_vec2.clear();

  /**
  *  That's it, the tutorial is completed.
  **/
  std::cout << "!!!! TUTORIAL COMPLETED SUCCESSFULLY !!!!" << std::endl;

  return EXIT_SUCCESS;
}

